#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <iostream>
#include <cstdlib>

#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;
int num_bins;
double bin_size;
int* sorted_parts; // separate array recording id of particles sorted by bin index
int* bin_start_idx; // start idx of the bins in sorted_parts
int* dynamic_assign_idx; // a temporal array for updating sorted_parts

// get bin_start_idx
__global__ void update_bin_start_idx(particle_t* particles, int* bin_start_idx, double bin_size, int num_bins, int num_parts) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    int bx = (int)floor(particles[tid].x / bin_size);
    int by = (int)floor(particles[tid].y / bin_size);
    int bin_index = bx + by * num_bins;
    if (bin_index < 0 || bin_index >= num_bins * num_bins) {
        printf("Thread %d calculated out-of-bounds bin_index: %d\n", tid, bin_index);
        return;
    }
    atomicAdd(bin_start_idx + bin_index, 1);
}

// assign particles to bins based on their corrent position, put their id / ori index in sorted_parts
__global__ void update_sorted_parts(particle_t* particles, int* sorted_parts, int* dynamic_assign_idx, double bin_size, int num_bins, int num_parts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;
    int bx = particles[tid].x / bin_size;
    int by = particles[tid].y / bin_size;
    int bin_index = bx + by * num_bins;
    
    int write_index = atomicAdd(&dynamic_assign_idx[bin_index], 1);
    sorted_parts[write_index] = tid;
}

static void gpu_init_arrays(int num_parts, int num_bins) {
    // Initialize sorted_parts, bin_start_idx and dynamic_assign_idx
    hipMalloc((void**)& sorted_parts, num_parts * sizeof(int));
    hipMalloc((void**)& bin_start_idx, num_bins * num_bins * sizeof(int));
    hipMalloc((void**)& dynamic_assign_idx, num_bins * num_bins * sizeof(int));
}

static void gpu_clear_arrays() {
    // Free memory allocated for sorted_parts, bin_start_idx and dynamic_assign_idx
    hipFree(sorted_parts);
    hipFree(bin_start_idx);
    hipFree(dynamic_assign_idx);
    
    // hipMalloc(bin_start_idx);
    // hipMalloc(dynamic_assign_idx);
}

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particles[tid].ax = particles[tid].ay = 0;

    
    for (int j = 0; j < num_parts; j++) {
        apply_force_gpu(particles[tid], particles[j]);
    }
}

// O(N) Code compute_forces_gpu() function
__global__ void compute_forces_gpu_ON(particle_t* particles, int num_parts, int* sorted_parts, int* bin_start_idx, double bin_size, int num_bins) {
    // Get the thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cur_bin_end_idx; 
    if (tid >= num_parts)
        return;

    particles[tid].ax = 0;
    particles[tid].ay = 0;
    int bx = particles[tid].x / bin_size;
    int by = particles[tid].y / bin_size;
    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            int nbx = bx + dx;
            int nby = by + dy;

            // Check if the neighboring bin is valid
            if (nbx >= 0 && nbx < num_bins && nby >= 0 && nby < num_bins) {
                int neighbor_bin_index = nbx + nby * num_bins;
                // find start_idx and end_idx of the neighbor bin in sorted_parts
                int cur_bin_start_idx = bin_start_idx[neighbor_bin_index];
                if (neighbor_bin_index + 1 < num_bins * num_bins) {
                    cur_bin_end_idx = bin_start_idx[neighbor_bin_index + 1];
                } else {
                    cur_bin_end_idx = num_parts;
                }

                // only compute force if the neighbor bin is not empty
                if (cur_bin_start_idx < cur_bin_end_idx) {
                    // Iterate over particles in the neighboring bin
                    for (int i =  cur_bin_start_idx; i < cur_bin_end_idx; ++i) {
                        apply_force_gpu(particles[tid], particles[sorted_parts[i]]);
                    }
                }
            }
        }
    }
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    // NOTE: This means that blks * NUM_THREADS >= num_parts, e.g. each particle
    // would have 1 thread to do computation.
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    // Initialize bin size and count (note on x and y axis 
    // these two numbers should be the same)
    num_bins = (int)floor(size / cutoff);
    bin_size = size / num_bins;

    std::cout << "num_bins " << num_bins << std::endl;
    std::cout << "bin_size " << bin_size << std::endl;

    gpu_init_arrays(num_parts, num_bins);

    std::cout << "Arrays initialized successfully." << std::endl;

    for (int i = 0; i < num_parts; ++i) {
        std::cout << "Particle " << i << std::endl;
        int bx = (int)floor(parts[i].x / bin_size);
        int by = (int)floor(parts[i].y / bin_size);
        int bin_index = bx + by * num_bins;
        std::cout << "Particle " << i << "has bx " << bx << "by " << by << "bin_index" << bin_index << std::endl;
    }
    exit();

    // This should update bin_start_idx to contain number of particles at each bin
    update_bin_start_idx<<<blks, NUM_THREADS>>>(parts, bin_start_idx, bin_size, num_bins, num_parts);
    
    std::cout << "update_bin_start_idx " << bin_start_idx[0] << std::endl;

    // in-place prefix sum
    thrust::exclusive_scan(bin_start_idx, bin_start_idx + num_bins * num_bins, bin_start_idx);
    
    std::cout << "prefix sum " << bin_start_idx[1] << std::endl;

    // copy data from bin_start_idx to dynamic_assign_idx
    hipMemcpy(dynamic_assign_idx, bin_start_idx, num_bins * num_bins * sizeof(int), hipMemcpyDeviceToDevice);
    
    std::cout << "copy " << dynamic_assign_idx[1] << std::endl;

    // get sorted_parts based on dynamic_assign_idx
    update_sorted_parts<<<blks, NUM_THREADS>>>(parts, sorted_parts, dynamic_assign_idx, bin_size, num_bins, num_parts);

    std::cout << "sorted_parts " << sorted_parts[dynamic_assign_idx[1]] << std::endl;
    
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    // compute the number of blocks needed
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    // Compute forces
    compute_forces_gpu_ON<<<blks, NUM_THREADS>>>(parts, num_parts, sorted_parts, bin_start_idx, bin_size, num_bins);
    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
    // Clear array from this iter
    gpu_clear_arrays();
    // Get new empty arrays for next iter
    gpu_init_arrays(num_parts, num_bins);
    // This should update bin_start_idx to contain number of particles at each bin
    update_bin_start_idx<<<blks, NUM_THREADS>>>(parts, bin_start_idx, bin_size, num_bins, num_parts);
    // in-place prefix sum
    thrust::exclusive_scan(bin_start_idx, bin_start_idx + num_bins * num_bins, bin_start_idx);
    // copy data from bin_start_idx to dynamic_assign_idx
    hipMemcpy(dynamic_assign_idx, bin_start_idx, num_bins * num_bins * sizeof(int), hipMemcpyDeviceToDevice);
    // get sorted_parts based on dynamic_assign_idx
    update_sorted_parts<<<blks, NUM_THREADS>>>(parts, sorted_parts, dynamic_assign_idx, bin_size, num_bins, num_parts);

}
